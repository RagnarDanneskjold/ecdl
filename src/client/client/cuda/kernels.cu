#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include "Fp.cu"
#include <stdio.h>

#define NUM_R_POINTS 32 // Must be a power of 2
#define R_POINT_MASK (NUM_R_POINTS - 1)

#define THREAD_ID (blockDim.x * blockIdx.x + threadIdx.x)
#define NUM_THREADS (gridDim.x * blockDim.x)

/**
 * Bit mask for identifying distinguished points
 */
__constant__ unsigned int _MASK[ 2 ];

/**
 * The X coordinates of the R points
 */
__constant__ unsigned int _rx[ 10 * NUM_R_POINTS ];

/**
 * The Y coordinates of the Y points
 */
__constant__ unsigned int _ry[ 10 * NUM_R_POINTS ];

/**
 * Shared memory to hold the R points
 */
__shared__ unsigned int _shared_rx[ 10 * NUM_R_POINTS ];
__shared__ unsigned int _shared_ry[ 10 * NUM_R_POINTS ];


/**
 * Point at infinity
 */
__device__ unsigned int _pointAtInfinity[10] = { 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff,
                                                 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff};

/**
 * Reads Rx[i] from shared memory
 */
template<int N> __device__ void getRX(int index, unsigned int *rx)
{
    for(int i = 0; i < N; i++) {
        rx[i] = _shared_rx[ 32 * i + index ];
    }
}

/**
 * Reads Ry[i] from shared memory
 */
template<int N> __device__ void getRY(int index, unsigned int *ry)
{
    for(int i = 0; i < N; i++) {
        ry[ i ] = _shared_ry[ 32 * i + index ];
    }
}


/**
 * Reads Rx and Ry from constant memory and writes them to shared memory
 */
__device__ void initSharedMem(unsigned int len)
{
    if( threadIdx.x == 0) {
        for(int i = 0; i < len; i++) {
            for(int j = 0; j < 32; j++) {
                _shared_rx[i * 32 + j] = _rx[ len * j + i];
                _shared_ry[i * 32 + j] = _ry[ len * j + i];
            }
        }
    }
    __syncthreads();
}

template<int N> __device__ void doMultiplyStep( const unsigned int *aMultiplier, const unsigned int *bMultiplier,
                                  const unsigned int *gx, const unsigned int *gy,
                                  const unsigned int *qx, const unsigned int *qy,
                                  const unsigned int *gqx, const unsigned int *gqy,
                                  unsigned int *xAra, unsigned int *yAra,
                                  unsigned int *diffBuf, unsigned int *chainBuf,
                                  int step, int pointsInParallel, int idx)
{
    int stride = gridDim.x * blockDim.x;

    unsigned int product[N] = {0};
    product[0] = 1;

    unsigned int mask = 1 << (step % 32);
    int word = step / 32;
  
    // To compute (Px - Qx)^-1, we multiply together all the differences and then perfom
    // a single inversion. After each multiplication we need to store the product.
    for(int i = 0; i < pointsInParallel; i++) {
        int offset = stride * i;
        unsigned int bpx[N];
        unsigned int x[N];

        readBigInt<N>(xAra, idx + offset, x);
        unsigned int diff[N];

        // For point at infinity we set the difference as 2 so the math still
        // works out 
        unsigned int am = readBigIntWord<N>( aMultiplier, idx + offset, word );
        unsigned int bm = readBigIntWord<N>( bMultiplier, idx + offset, word );

        if( (am | bm) & mask == 0 || equalTo<N>(x, _pointAtInfinity) ) {
            zero<N>(diff);
            diff[0] = 2;
        } else {
            if( (am & ~bm) & mask ) {
                copy<N>(&gx[step *N], bpx);
            } else if( (~am & bm) & mask) {
                copy<N>(&qx[step *N], bpx);
            } else {
                copy<N>(&gqx[step *N], bpx);
            }
            subModP<N>(x, bpx, diff);
        }

        writeBigInt<N>(diffBuf, idx + offset, diff);

        multiplyModP<N>(product, diff, product);
        writeBigInt<N>(chainBuf, idx + offset, product);
    }

    // Compute the inverse
    unsigned int inverse[N];
    inverseModP<N>(product, inverse);

    // Multiply by the products stored perviously so that they are canceled out
    for(int i = pointsInParallel - 1; i >= 0; i--) {
        // Get the inverse of the last difference by multiplying the inverse of the product of all the differences
        // with the product of all but the last difference
        unsigned int invDiff[N];
        if( i >= 1) {
            unsigned int tmp[N];
            readBigInt<N>(chainBuf, idx + stride * (i-1), tmp);
            multiplyModP<N>(inverse, tmp, invDiff);

            // Cancel out the last difference
            readBigInt<N>(diffBuf, idx + stride * i, tmp);
            multiplyModP<N>(inverse, tmp, inverse);
        } else {
            copy<N>(inverse, invDiff);
        }
      
        int offset = stride * i;
        unsigned int am = readBigIntWord<N>( aMultiplier, idx + offset, word );
        unsigned int bm = readBigIntWord<N>( bMultiplier, idx + offset, word );

        if( (am & mask) != 0 || (bm & mask) != 0 ) {
            unsigned int px[N];
            unsigned int py[N];
            unsigned int bpx[N];
            unsigned int bpy[N];
          
            // Select G, Q, or G+Q 
            if( (am & ~bm) & mask ) {
                copy<N>(&gx[step *N], bpx);
                copy<N>(&gy[step *N], bpy);
            } else if( (~am & bm) & mask) {
                copy<N>(&qx[step *N], bpx);
                copy<N>(&qy[step *N], bpy);
            } else {
                copy<N>(&gqx[step *N], bpx);
                copy<N>(&gqy[step *N], bpy);
            }

            // Load the current point
            readBigInt<N>(xAra, idx + offset, px);
            readBigInt<N>(yAra, idx + offset, py);

            if( equalTo<N>( px, _pointAtInfinity ) ) {
                writeBigInt<N>(xAra, idx + offset, bpx);
                writeBigInt<N>(yAra, idx + offset, bpy);
            } else {
                unsigned int s[N];
                unsigned int rx[N];
                unsigned int s2[N];

                // s = Py - Qy / Px - Qx
                subModP<N>(py, bpy, s);
                multiplyModP<N>(invDiff, s, s);
                squareModP<N>(s, s2);

                // Rx = s^2 - Px - Qx
                subModP<N>(s2, px, rx);
                subModP<N>(rx, bpx, rx);

                // Ry = -Py + s(Px - Rx)
                unsigned int k[N];
                subModP<N>(px, rx, k);
                multiplyModP<N>(k, s, k);
                unsigned int ry[N];

                subModP<N>(k, py, ry);

                writeBigInt<N>(xAra, idx + offset, rx);
                writeBigInt<N>(yAra, idx + offset, ry);
            }
        }
    }
}

/**
 * Based on the bit values of a and b, G, Q, or (G+Q) will be added
 */
__global__ void startingPointGenKernel( const unsigned int *a, const unsigned int *b,
                                       const unsigned int *gx, const unsigned int *gy,
                                       const unsigned int *qx, const unsigned int *qy,
                                       const unsigned int *gqx, const unsigned int *gqy,
                                       unsigned int *rx, unsigned int *ry,
                                       unsigned int *diffBuf, unsigned int *chainBuf,
                                       int step, unsigned int totalPoints, unsigned int pointsInParallel)
{
    int stride = NUM_THREADS * pointsInParallel;

    initFp();
    initSharedMem(_PWORDS);

    for(int idx = THREAD_ID; idx < totalPoints; idx += stride) {

        switch(_PWORDS) {
            case 2:
            doMultiplyStep<2>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break; 
            case 3:
            doMultiplyStep<3>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break;
            case 4:
            doMultiplyStep<4>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break;
            case 5:
            doMultiplyStep<5>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break;
            case 6:
            doMultiplyStep<6>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break;
            case 7:
            doMultiplyStep<7>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break;
            case 8:
            doMultiplyStep<8>( a, b, gx, gy, qx, qy, gqx, gqy, rx, ry, diffBuf, chainBuf, step, pointsInParallel, idx);
            break;
        }
    }
}

/**
 * Resets all points on the device to the point at infinity
 */
__device__ void resetPointsFunc(unsigned int *rx, unsigned int *ry, int totalPoints)
{
    // Reset all points to the identity element
    unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;

    for(int i = threadId; i < totalPoints; i += stride)
    {
        writeBigInt(rx, i, _pointAtInfinity, _PWORDS);
        writeBigInt(ry, i, _pointAtInfinity, _PWORDS);
    }
}

/**
 * Kernel to reset all points on the device to the point at infinity
 */
__global__ void resetPointsKernel( unsigned int *rx, unsigned int *ry, int count)
{
    resetPointsFunc(rx, ry, count);
}


/**
 * Sets the number of distinguished bits to look for
 */
hipError_t setNumDistinguishedBits(unsigned int dBits)
{
    unsigned int mask[2] = {0xffffffff, 0xffffffff};
    if(dBits > 32) {
        mask[ 1 ] >>= (32 - (dBits - 32));
    } else {
        mask[ 0 ] >>= (32 - dBits);
        mask[ 1 ] = 0;
    }
    return hipMemcpyToSymbol(HIP_SYMBOL(_MASK), mask, sizeof(mask), 0, hipMemcpyHostToDevice);
}


/**
 * Subtract 2 from a big integer
 */
static void sub2(const unsigned int *a, unsigned int *c, int len)
{
    unsigned int borrowOut = 0;
    unsigned int borrowIn = 2;

    for( int i = 0; i < len; i++ ) {
      
        unsigned int d = a[ i ] - borrowIn;

        if(d > a[i]) {
            borrowOut = 1;
        } else {
            borrowOut = 0;
        }

        borrowIn = borrowOut;
        c[i] = d;
    }
}

/**
 * Shift a big integer left by n bits
 */
static void shiftLeft(const unsigned int *a, int n, unsigned int *c, int len)
{
    unsigned int out = 0;
    unsigned int in = 0;
    for(int i = 0; i < len; i++) {
        out = a[i] >> (32 - n);
        c[i] = a[i] << n;
        c[i] |= in;
        in = out;
    }
}

/**
 * Add two big integers
 */
static void addInt(const unsigned int *a, const unsigned int *b, unsigned int *c, int len)
{
    unsigned int carryIn = 0;
    unsigned int carryOut = 0;
    for(int i = 0; i < len; i++) {

        unsigned int s = a[i] + b[i];

        if(s < a[i]) {
            carryOut = 1;
        } else {
            carryOut = 0;
        }

        s += carryIn;

        carryIn = carryOut;

        c[i] = s;
    }
}

/**
 * Set parameters for the prime field library
 */
hipError_t setFpParameters(const unsigned int *pPtr, unsigned int pBits, const unsigned int *mPtr, unsigned int mBits)
{
    hipError_t hipError_t = hipSuccess;
    unsigned int pWords = (pBits + 31) / 32;
    unsigned int mWords = (mBits + 31) / 32;
    unsigned int p2Words = (pBits + 1 + 31) / 32;
    unsigned int p3Words = (pBits + 2 + 31) / 32;

    unsigned int p[10] = {0};
    unsigned int pTimes2[10] = {0};
    unsigned int pTimes3[10] = {0};
    unsigned int pMinus2[10] = {0};

    // copy p into buffer
    for(unsigned int i = 0; i < pWords; i++) {
        p[i] = pPtr[i];
    }

    // compute p - 2
    sub2(p, pMinus2, 10);

    // compute 2 * p
    shiftLeft(p, 1, pTimes2, 10);

    // compute 3 * p
    addInt(p, pTimes2, pTimes3, 10);

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_P_CONST), p, sizeof(unsigned int)*pWords, 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }
    
    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_PMINUS2_CONST), pMinus2, sizeof(unsigned int)*pWords, 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }
    
    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_M_CONST), mPtr, sizeof(unsigned int)*mWords, 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_PBITS_CONST), &pBits, sizeof(pBits), 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_2P_CONST), pTimes2, sizeof(unsigned int) * p2Words, 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_3P_CONST), pTimes3, sizeof(unsigned int) * p3Words, 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_PWORDS), &pWords, sizeof(unsigned int), 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_MWORDS), &mWords, sizeof(unsigned int), 0, hipMemcpyHostToDevice);
    if(hipError_t != hipSuccess) {
        goto end;
    }


    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_MBITS_CONST), &mBits, sizeof(mBits), 0, hipMemcpyHostToDevice);
    
end:
    return hipError_t;
}


/**
 * Initialize device parameters
 */
hipError_t initDeviceParams(const unsigned int *p, unsigned int pBits, const unsigned int *m, unsigned int mBits, unsigned int dBits)
{
    hipError_t hipError_t = setFpParameters(p, pBits, m, mBits);
    if(hipError_t != hipSuccess) {
        goto end;
    }

    hipError_t = setNumDistinguishedBits(dBits);

end:
    return hipError_t;
}

/**
 * Copy a, b, Rx and Ry to constant memory
 */
hipError_t copyRPointsToDevice(const unsigned int *rx, const unsigned int *ry, int length, int count)
{
    hipError_t hipError_t = hipSuccess;
    size_t size = sizeof(unsigned int) * length * count;

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL( _rx), rx, size, 0, hipMemcpyHostToDevice );
    if( hipError_t != hipSuccess ) {
        goto end;
    }

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL( _ry), ry, size, 0, hipMemcpyHostToDevice );
    if( hipError_t != hipSuccess ) {
        goto end;
    }

end:
    return hipError_t;
}

hipError_t multiplyAddG( int blocks, int threads,
                          const unsigned int *a, const unsigned int *b,
                          const unsigned int *gx, const unsigned int *gy,
                          const unsigned int *qx, const unsigned int *qy,
                          const unsigned int *gqx, const unsigned int *gqy,
                          unsigned int *x, unsigned int *y,
                          unsigned int *diffBuf, unsigned int *chainBuf,
                          unsigned int totalPoints, unsigned int pointsInParallel, int step)
{
    startingPointGenKernel<<<blocks, threads>>>(a, b, gx, gy, qx, qy, gqx, gqy, x, y, diffBuf, chainBuf, step, totalPoints, pointsInParallel);
    return hipDeviceSynchronize();
}

/**
 * Reset all points to point at infinity
 */
hipError_t resetPoints( int blocks, int threads, unsigned int *rx, unsigned int *ry, int count )
{
    resetPointsKernel<<<blocks, threads>>>(rx, ry, count);
    return hipDeviceSynchronize();
}

void __device__ cuPrintBigInt(const unsigned int *x, int len)
{
    for(int i = 0; i < len; i++) {
        printf("%x ", x[i]);
    }
    printf("\n");
}


template<int N> __device__ void doStepMulti(
                            unsigned int *xAra,
                            unsigned int *yAra,
                            unsigned int *diffBuf,
                            unsigned int *chainBuf,
                            unsigned int *pointFound,
                            unsigned int *sectionFlags,
                            unsigned int *flags,
                            unsigned int pointsInParallel,
                            unsigned int idx) {

    int stride = gridDim.x * blockDim.x;

    // Initalize to 1
    unsigned int product[N] = {0};
    product[0] = 1;

    // Reset the point found flag
    if( blockIdx.x == 0 && threadIdx.x == 0 ) {
        *pointFound = 0;
    }

    // Multiply differences together
    for(int i = 0; i < pointsInParallel; i++) {
        unsigned int offset = stride * i;
        unsigned int x[N];
        readBigInt<N>(xAra, idx + offset, x);
        unsigned int rIdx = x[0] & R_POINT_MASK;

        unsigned int diff[N];
        unsigned int rx[N];
        getRX<N>(rIdx, rx);
        subModP<N>(x, rx, diff);

        writeBigInt<N>(diffBuf, idx + offset, diff);

        multiplyModP<N>(product, diff, product);
        writeBigInt<N>(chainBuf, idx + offset, product);
    }

    // Compute inverse
    unsigned int inverse[N];
    inverseModP<N>(product, inverse);

    // Extract inverse of the differences
    for(int i = pointsInParallel - 1; i >= 0; i--) {
        unsigned int offset = stride * i; 

        // Get the inverse of the last difference by multiplying the inverse of the product of all the differences
        // with the product of all but the last difference
        unsigned int invDiff[N];

        if(i >= 1) {
            unsigned int tmp[N];
            readBigInt<N>(chainBuf, idx + stride * (i-1), tmp);
            multiplyModP<N>(inverse, tmp, invDiff);

            // Cancel out the last difference
            readBigInt<N>(diffBuf, idx + stride * i, tmp);
            multiplyModP<N>(inverse, tmp, inverse);

        } else {
            copy<N>(inverse, invDiff);
        }
        
        unsigned int px[N];
        unsigned int py[N];

        readBigInt<N>(xAra, idx + offset, px);
        readBigInt<N>(yAra, idx + offset, py);

        unsigned int rIdx = px[0] & R_POINT_MASK;
        unsigned int s[N];
        unsigned int s2[N];

        // s^2 = (Py - Qy / Px - Qx)^2
        unsigned int ry[N];
        getRY<N>(rIdx, ry);
        subModP<N>(py, ry, s);
        multiplyModP<N>(s, invDiff, s);
        squareModP<N>(s, s2);

        // Rx = s^2 - Px - Qx
        unsigned int newX[N];
        subModP<N>(s2, px, newX);

        unsigned int rx[N];
        getRX<N>(rIdx, rx);
        subModP<N>(newX, rx, newX);

        // Ry = -Py + s(Px - Rx)
        unsigned int k[N];
        subModP<N>(px, newX, k);
        multiplyModP<N>(k, s, k);
        unsigned int newY[N];
        subModP<N>(k, py, newY);

        // Write resul to memory
        writeBigInt<N>(xAra, idx + offset, newX);
        writeBigInt<N>(yAra, idx + offset, newY);
       
        // Check for distinguished point, set flag if found
        if(((newX[ 0 ] & _MASK[ 0 ]) == 0) && ((newX[ 1 ] & _MASK[ 1 ]) == 0)) {
            int section = (idx + offset) / 32;
            sectionFlags[section] = 1;
            *pointFound = 1;
            flags[idx + offset] = 1;
        }
    }
}

__global__ void doStepKernel( unsigned int *xAra,
                              unsigned int *yAra,
                              unsigned int *diffBuf,
                              unsigned int *chainBuf,
                              unsigned int *pointFound,
                              unsigned int *blockFlags,
                              unsigned int *flags,
                              unsigned int totalPoints,
                              unsigned int pointsInParallel )
{
    int stride = NUM_THREADS * pointsInParallel;

    // Initialize shared memory constants
    initFp();
    initSharedMem(_PWORDS);

    for(int idx = THREAD_ID; idx < totalPoints; idx += stride) {

        switch(_PWORDS) {
            case 2:
            doStepMulti<2>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
            case 3:
            doStepMulti<3>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
            case 4:
            doStepMulti<4>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
            case 5:
            doStepMulti<5>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
            case 6:
            doStepMulti<6>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
            case 7:
            doStepMulti<7>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
            case 8:
            doStepMulti<8>(xAra, yAra, diffBuf, chainBuf, pointFound, blockFlags, flags, pointsInParallel, idx);
            break;
        }
    }
}

hipError_t initDeviceConstants(unsigned int numPoints)
{
    hipError_t hipError_t = hipSuccess;

    hipError_t = hipMemcpyToSymbol(HIP_SYMBOL(_NUM_POINTS), &numPoints, sizeof(unsigned int), 0, hipMemcpyHostToDevice);

    if(hipError_t != hipSuccess) {
        goto end;
    }

end:
    return hipSuccess;
}

hipError_t cudaDoStep( int blocks,
                    int threads,
                    unsigned int *rx,
                    unsigned int *ry,
                    unsigned int *diffBuf,
                    unsigned int *chainBuf,
                    unsigned int *pointFound,
                    unsigned int *pointIndex,
                    unsigned int *flags,
                    unsigned int totalPoints,
                    unsigned int pointsInParallel)
{
    doStepKernel<<<blocks, threads>>>(rx,
                                      ry,
                                      diffBuf,
                                      chainBuf,
                                      pointFound,
                                      pointIndex,
                                      flags,
                                      totalPoints,
                                      pointsInParallel);
    return hipDeviceSynchronize();
}
